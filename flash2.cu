#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__
void forward_kernel(const float* Q, const float* K, const float* V, const int seqlen, const int headdim,
                    const int Tc, const int Tr, const int Bc, const int Br, const float softmax_scale, 
		    const int nhead, float* O) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int head_idx = blockIdx.y;
    int bz = blockIdx.z;

    // Offset into Q,K,V,O,l,m - different for each batch and head
    int qkv_offset = (bx * nhead * seqlen * headdim) + (head_idx * seqlen * headdim);

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size_qo = Br * headdim;  // size of Qi, Oi
    int tile_size_kv = Bc * headdim;  // size of Kj, Vj
    float* Qi = sram;
    float* Oi = &sram[tile_size_qo];
    float* Kj = &sram[tile_size_qo * 2];
    float* Vj = &sram[tile_size_qo * 2 + tile_size_kv];
    float* S = &sram[tile_size_qo * 2 + tile_size_kv * 2];

    // Load Qi to SRAM
    for (int x = 0; x < headdim; x++) {
        Qi[(tx * headdim) + x] = Q[qkv_offset + (tile_size_qo * bz) + (tx * headdim) + x];
        Oi[(tx * headdim) + x] = 0; // zero
    }
    
    float row_m_prev = -INFINITY;
    float row_l_prev = 0;
    float row_m_new, row_l_new;

    for (int j = 0; j < Tc; j++)  {
        // Load Kj, Vj to SRAM
        for (int x = 0; x < headdim; x++) {
            Kj[(tx * headdim) + x] = K[qkv_offset + (tile_size_kv * j) + (tx * headdim) + x];
            Vj[(tx * headdim) + x] = V[qkv_offset + (tile_size_kv * j) + (tx * headdim) + x];
        }
        // S = QK^T, row_m = rowmax(S)
        float row_m = -INFINITY;

	    int not_mask = 1;
        for (int y = 0; y < Bc; y++) {
            float sum = 0;
            for (int x = 0; x < headdim; x++) {
                sum += Qi[(tx * headdim) + x] * Kj[(y * headdim) + x];
            }
            sum *= softmax_scale;
            S[(Bc * tx) + y] = (~not_mask & (bz * Br + tx < j * Bc + y)) ? -INFINITY : sum;

            if (sum > row_m) row_m = sum;
        }
        
        // Compute new m
        row_m_new = max(row_m_prev, row_m);

        // P = exp(S - row_m), row_l = rowsum(P)
        float row_l = 0;
        for (int y = 0; y < Bc; y++) {
            S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m_new);
            row_l += S[(Bc * tx) + y];
        }

        // Compute l
        row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + row_l;

        // Write O, l, m to HBM
        for (int x = 0; x < headdim; x++) {
            float pv = 0;  // Pij * Vj
            for (int y = 0; y < Bc; y++) {
                pv += S[(Bc * tx) + y] * Vj[(y * headdim) + x];
            }
            Oi[(tx * headdim) + x] = (__expf(row_m_prev - row_m_new)) * Oi[(tx * headdim) + x] + pv;
        }

        // Update l, m
        row_l_prev = row_l_new;
        row_m_prev = row_m_new;
    }
    for (int x = 0; x < headdim; x++) {
        O[qkv_offset + (tile_size_qo * bz) + (tx * headdim) + x] = 1 / row_l_new * Oi[(tx * headdim) + x];
    }
}

torch::Tensor forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
    // TODO: determine Bc, Br dynamically
    const int Bc = 32;
    const int Br = 32;

    const int nbatch = Q.size(0); 
    const int nhead = Q.size(1);
    const int seqlen = Q.size(2);
    const int headdim = Q.size(3);

    const int Tc = ceil((float) seqlen / Bc);
    const int Tr = ceil((float) seqlen / Br);
    const float softmax_scale = 1.0 / sqrt(headdim);

    // Initialize O, l, m to HBM
    auto O = torch::zeros_like(Q);
    torch::Device device(torch::kCUDA);

    // Calculate SRAM size needed per block
    const int sram_size = (2 * Br * headdim * sizeof(float)) + (2 * Bc * headdim * sizeof(float)) + (Bc * Br * sizeof(float));
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \\n", max_sram_size, sram_size);

    dim3 grid_dim(nbatch, nhead, Tr);  // batch_size x num_heads x Tr
    dim3 block_dim(Bc);  // Bc threads per block

    forward_kernel<<<grid_dim, block_dim, sram_size>>>(
        Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
        seqlen, headdim, Tc, Tr, Bc, Br, softmax_scale, nhead, O.data_ptr<float>()
    );
    return O;
}